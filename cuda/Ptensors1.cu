#include "hip/hip_runtime.h"
/*
This file is part of ptens, a C++/CUDA library for permutation 
equivariant message passing. 
 
Copyright (c) 2023, Imre Risi Kondor

This source code file is subject to the terms of the noncommercial 
license distributed with cnine in the file LICENSE.TXT. Commercial 
use is prohibited. All redistributed versions of this file (in 
original or modified form) must retain this copyright notice and 
must be accompanied by a verbatim copy of the license. 
*/

#ifndef _Ptensors1_cu
#define _Ptensors1_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/tuple.h>

#include "Ptens_base.hpp"
#include "RtensorPack.hpp"
#include "AindexPack.hpp"


__forceinline__ __device__ int load_indices(int* ix, const int* xiarr, const int* xidir, const int q){
  int offs=xidir[2*q];
  int n=xidir[2*q+1];
  int t=threadIdx.x;
  if(t<n){
    ix[t]=xiarr[offs+t];
  }
  return n-1;
}


// ---- mprod ------------------------------------------------------------------------------------------------


__global__ void Ptensors1_add_mprod(float* rarr, const int* rdir, const float* xarr, const int* xdir, const float* yarr){
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=xdir[3*q+1];
  const int ncx=xdir[3*q+2];
  const int ncr=rdir[3*q+2];

  const float* x=xarr+xdir[3*q];
  float* r=rarr+rdir[3*q];

  for(int i=0; i<k; i++){
    float t=0;
    const float* xrow=x+i*ncx;
    const float* ycol=yarr+c;
    for(int j=0; j<ncx; j++)
      t+=xrow[j]*ycol[j*ncr];
    r[i*ncr+c]+=t;
  }
}


/*
__global__ void Ptensors1_add_mprod_back0(float* rarr, const int* rdir, const float* rarr, const int* rdir, const float* yarr){
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=xdir[3*q+1];
  const int ncx=xdir[3*q+2];
  const int ncr=rdir[3*q+2];

  const float* x=xarr+xdir[3*q];
  const float* r=rarr+rdir[3*q];

  for(int i=0; i<k; i++){
    float t=0;
    float* xrow=x+i*ncx;
    float* yrow=yarr+c*ncx;
    for(int j=0; j<ncx; j++)
      t+=xrow[j]*yrow[j];
    r[i*ncr+c]+=t;
  }
}


__global__ void Ptensors1_add_mprod_back1(float* rarr, const int* rdir, const float* rarr, const int* rdir, const float* yarr){
}
*/

// ---- Reduce -----------------------------------------------------------------------------------------------


__global__ void Ptensors1_reduce0_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=xdir[3*q+1];
  const int nc=xdir[3*q+2];
  //if(c>=nc) return;

  const float* x=xarr+xdir[3*q]+c;
  float t=0;
  for(int i=0; i<k; i++)
    t+=x[i*nc];
  rarr[rdir[2*q]+c]+=t;
}


__global__ void Ptensors1_reduce0n_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=xdir[3*q+1];
  const int nc=xdir[3*q+2];
  //if(c>=nc) return;

  const float* x=xarr+xdir[3*q]+c;
  float t=0;
  for(int i=0; i<k; i++)
    t+=x[i*nc];
  rarr[rdir[2*q]+c]+=t/k;
}


__global__ void Ptensors1_reduce0_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const int* xiarr, const int* xidir, const int n){
  extern __shared__ unsigned char _shared[]; 
  int* ix=reinterpret_cast<int*>(_shared);
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=load_indices(ix,xiarr,xidir,q);
  __syncthreads();
  const int nc=xdir[2];
  if(c>=n) return;

  const float* x=xarr+xdir[3*ix[0]]+c;
  float t=0;
  for(int i=0; i<k; i++)
    t+=x[ix[i+1]*nc];
  rarr[rdir[2*q]+c]+=t;
}


__global__ void Ptensors1_reduce0n_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const int* xiarr, const int* xidir, const int n){
  extern __shared__ unsigned char _shared[]; 
  int* ix=reinterpret_cast<int*>(_shared);
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=load_indices(ix,xiarr,xidir,q);
  __syncthreads();
  const int nc=xdir[2];
  if(c>=n) return;

  const float* x=xarr+xdir[3*ix[0]]+c;
  float t=0;
  for(int i=0; i<k; i++)
    t+=x[ix[i+1]*nc];
  rarr[rdir[2*q]+c]+=t/k;
}


__global__ void Ptensors1_reduce1_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=xdir[3*q+1];
  const int nc=xdir[3*q+2];
  const int rnc=rdir[3*q+2];
  //if(c>=nc) return;

  const float* x=xarr+xdir[3*q]+c;
  float* r=rarr+rdir[3*q]+c;
  for(int i=0; i<k; i++)
    r[i*rnc]+=x[i*nc];
}


__global__ void Ptensors1_reduce1_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const int* xiarr, const int* xidir, const int n){
  extern __shared__ unsigned char _shared[]; 
  int* ix=reinterpret_cast<int*>(_shared);
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=load_indices(ix,xiarr,xidir,q);
  __syncthreads();
  const int nc=xdir[2];
  const int rnc=rdir[2];
  if(c>=n) return;

  const float* x=xarr+xdir[3*ix[0]]+c;
  float* r=rarr+rdir[3*q]+c;
  for(int i=0; i<k; i++)
    r[i*rnc]+=x[ix[i+1]*nc];
}


// ---- Broadcast --------------------------------------------------------------------------------------------


__global__ void Ptensors1_broadcast0_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir){
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=xdir[3*q+1];
  const int nc=xdir[3*q+2];
  const int rnc=rdir[2*q+1];
  if(c>=rnc) return;

  float* x=xarr+xdir[3*q]+c;
  const float t=rarr[rdir[2*q]+c];
  for(int i=0; i<k; i++)
    x[i*nc]+=t;
}


__global__ void Ptensors1_broadcast0n_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir){
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=xdir[3*q+1];
  const int nc=xdir[3*q+2];
  const int rnc=rdir[2*q+1];
  if(c>=rnc) return;

  float* x=xarr+xdir[3*q]+c;
  const float t=rarr[rdir[2*q]+c]/k;
  for(int i=0; i<k; i++)
    x[i*nc]+=t;
}


__global__ void Ptensors1_broadcast0_kernel(float* xarr, const int* xdir, const int* xiarr, const int* xidir, 
  const float* rarr, const int* rdir, const int* bmap){
  extern __shared__ unsigned char _shared[]; 
  int* ix=reinterpret_cast<int*>(_shared);

  const int b=blockIdx.x;
  const int c=threadIdx.x;
  const int boffs=bmap[3*b];
  const int N=bmap[3*b+1];
  const int target=bmap[3*b+2];
  //if(c==0) printf("%d target=%d\n",b,target);

  const int nc=xdir[2]; //xdir[3*target+2];
  const int rnc=rdir[1]; //xdir[3*target+2];


  float* x=xarr+xdir[3*target]+c;
  for(int s=0; s<N; s++){
    const int src=bmap[boffs+2*s];
    //if(c==0) printf("%d %d %d\n",b,s,src);
    const int k=load_indices(ix,xiarr,xidir,src);
    __syncthreads();
    if(c>=rnc) continue;
    float t=rarr[rdir[2*src]+c];
    //if(c==0) printf("%d %d %d %f\n",b,s,src,t);
    for(int i=0; i<k; i++){
      x[ix[i+1]*nc]+=t;
    }
  }

  return;
}


__global__ void Ptensors1_broadcast0n_kernel(float* xarr, const int* xdir, const int* xiarr, const int* xidir, 
  const float* rarr, const int* rdir, const int* bmap){
  extern __shared__ unsigned char _shared[]; 
  int* ix=reinterpret_cast<int*>(_shared);

  const int b=blockIdx.x;
  const int c=threadIdx.x;
  const int boffs=bmap[3*b];
  const int N=bmap[3*b+1];
  const int target=bmap[3*b+2];
  //if(c==0) printf("%d target=%d\n",b,target);

  const int nc=xdir[2]; //xdir[3*target+2];
  const int rnc=rdir[1]; //xdir[3*target+2];


  float* x=xarr+xdir[3*target]+c;
  for(int s=0; s<N; s++){
    const int src=bmap[boffs+2*s];
    //if(c==0) printf("%d %d %d\n",b,s,src);
    const int k=load_indices(ix,xiarr,xidir,src);
    __syncthreads();
    if(c>=rnc) continue;
    float t=rarr[rdir[2*src]+c]/k;
    //if(c==0) printf("%d %d %d %f\n",b,s,src,t);
    for(int i=0; i<k; i++){
      x[ix[i+1]*nc]+=t;
    }
  }

  return;
}


__global__ void Ptensors1_broadcast1_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir){
  const int q=blockIdx.x;
  const int c=threadIdx.x;
  const int k=xdir[3*q+1];
  const int nc=xdir[3*q+2];
  const int rnc=rdir[3*q+2];
  if(c>=rnc) return;

  float* x=xarr+xdir[3*q]+c;
  const float* r=rarr+rdir[3*q]+c;
  for(int i=0; i<k; i++)
    x[i*nc]+=r[i*rnc];
}


__global__ void Ptensors1_broadcast1_kernel(float* xarr, const int* xdir, const int* xiarr, const int* xidir, 
  const float* rarr, const int* rdir, const int* bmap){
  extern __shared__ unsigned char _shared[]; 
  int* ix=reinterpret_cast<int*>(_shared);
  const int b=blockIdx.x;
  const int c=threadIdx.x;
  const int boffs=bmap[3*b];
  const int N=bmap[3*b+1];
  const int target=bmap[3*b+2];
  //if(c==0) printf("target=%d\n",target);

  const int nc=xdir[2]; //xdir[3*target+2];
  const int rnc=rdir[2]; //xdir[3*target+2];

  float* x=xarr+xdir[3*target]+c;
  for(int s=0; s<N; s++){
    const int src=bmap[boffs+2*s];
    const int k=load_indices(ix,xiarr,xidir,src);
    __syncthreads();
    //if(c>=rnc) return;
    if(c>=rnc) continue; // changed 

    const float* r=rarr+rdir[3*src]+c;
    for(int i=0; i<k; i++){
      //if(c==0) printf("%d %d %d %d %d %d %d\n",src,target,i,ix[i+1],xdir[3*target],xdir[3*target+1],nc);
      x[ix[i+1]*nc]+=r[i*rnc];
    }
  }
}


// ---- Outer -----------------------------------------------------------------------------------------------


__global__ void Ptensors1_add_outer10_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const float* yarr, const int* ydir){
  const int q=blockIdx.x;
  const int xc=threadIdx.x;
  const int yc=threadIdx.y;
  const int rc=xc*ydir[2*q+1]+yc;
  const int k=xdir[3*q+1];
  const int nxc=xdir[3*q+2];
  const int nrc=rdir[3*q+2];

  float* r=rarr+rdir[3*q]+rc;
  const float* x=xarr+xdir[3*q]+xc;
  const float t=yarr[ydir[2*q]+yc];
  for(int i=0; i<k; i++)
    r[i*nrc]+=t*x[i*nxc];
}


__global__ void Ptensors1_add_outer10_back0_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir, const float* yarr, const int* ydir){
  const int q=blockIdx.x;
  const int xc=threadIdx.x;
  const int rc=xc*ydir[2*q+1];
  const int k=xdir[3*q+1];
  const int nxc=xdir[3*q+2];
  const int nyc=ydir[2*q+1];
  const int nrc=rdir[3*q+2];

  float* x=xarr+xdir[3*q]+xc;
  const float* r=rarr+rdir[3*q]+rc;
  const float* y=yarr+ydir[2*q];

  for(int i=0; i<k; i++){
    float t=0;
    for(int yc=0; yc<nyc; yc++)
      t+=r[i*nrc+yc]*y[yc];
    x[i*nxc]+=t;
  }
}


__global__ void Ptensors1_add_outer10_back1_kernel(float* yarr, const int* ydir, const float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int q=blockIdx.x;
  const int yc=threadIdx.x;
  const int k=xdir[3*q+1];
  const int nxc=xdir[3*q+2];
  const int nyc=ydir[2*q+1];
  const int nrc=rdir[3*q+2];

  float t=0;
  for(int i=0; i<k; i++){
    const float* x=xarr+xdir[3*q]+i*nxc;
    const float* r=rarr+rdir[3*q]+i*nrc+yc;
    for(int xc=0; xc<nxc; xc++)
      t+=r[nyc*xc]*x[xc];
  }
  yarr[ydir[2*q]+yc]+=t;
  
}

__global__ void Ptensors1_add_outer01_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const float* yarr, const int* ydir){
  const int q=blockIdx.x;
  const int xc=threadIdx.x;
  const int yc=threadIdx.y;
  const int rc=xc*ydir[3*q+2]+yc;
  const int k=ydir[3*q+1];
  //const int nxc=xdir[2*q+1];
  const int nyc=ydir[3*q+2];
  const int nrc=rdir[3*q+2];

  float* r=rarr+rdir[3*q]+rc;
  const float* y=yarr+ydir[3*q]+yc;
  const float t=xarr[xdir[2*q]+xc];
  for(int i=0; i<k; i++)
    r[i*nrc]+=t*y[i*nyc];
}


__global__ void Ptensors1_add_outer01_back0_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir, const float* yarr, const int* ydir){
  const int q=blockIdx.x;
  const int xc=threadIdx.x;
  const int rc=xc*ydir[3*q+2];
  const int k=ydir[3*q+1];
  //const int nxc=xdir[2*q+1];
  const int nyc=ydir[3*q+2];
  const int nrc=rdir[3*q+2];

 float t=0;
  for(int i=0; i<k; i++){
    const float* y=yarr+ydir[3*q]+i*nyc;
    const float* r=rarr+rdir[3*q]+i*nrc+rc;
    for(int yc=0; yc<nyc; yc++)
      t+=r[yc]*y[yc];
  }
  xarr[xdir[2*q]+xc]+=t;

}


__global__ void Ptensors1_add_outer01_back1_kernel(float* yarr, const int* ydir, const float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int q=blockIdx.x;
  const int yc=threadIdx.x;
  const int k=ydir[3*q+1];
  const int nxc=xdir[2*q+1];
  const int nyc=ydir[3*q+2];
  const int nrc=rdir[3*q+2];

  float* y=yarr+ydir[3*q]+yc;
  const float* r=rarr+rdir[3*q]+yc;
  const float* x=xarr+xdir[2*q];

  for(int i=0; i<k; i++){
    float t=0;
    for(int xc=0; xc<nxc; xc++)
      t+=r[i*nrc+xc*nyc]*x[xc];
    y[i*nyc]+=t;
  }
   
}


// -----------------------------------------------------------------------------------------------------------


namespace ptens{


  void Ptensors1_reduce0_cu(cnine::RtensorPackB& R, const cnine::RtensorPackB& x, 
    int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(R.size()==0) return;
    Ptensors1_reduce0_kernel<<<R.size(),n,0,stream>>>(R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev));
  }

  void Ptensors1_reduce0n_cu(cnine::RtensorPackB& R, const cnine::RtensorPackB& x, 
    int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(R.size()==0) return;
    Ptensors1_reduce0n_kernel<<<R.size(),n,0,stream>>>(R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev));
  }

  void Ptensors1_reduce0_cu(cnine::RtensorPackB& R, const cnine::RtensorPackB& x, const AindexPack& list, 
    int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(list.size()==0) return;
    const_cast<AindexPack&>(list).to_device(1);
    PTENS_ASSRT(list.dev==1);
    const int nthrd=cnine::roundup(std::max(n,list.max_nix()+1),32);
    Ptensors1_reduce0_kernel<<<list.size(),nthrd,(list.max_nix()+1)*4,stream>>>
      (R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),n);
  }

  void Ptensors1_reduce0n_cu(cnine::RtensorPackB& R, const cnine::RtensorPackB& x, const AindexPack& list, 
    int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(list.size()==0) return;
    const_cast<AindexPack&>(list).to_device(1);
    PTENS_ASSRT(list.dev==1);
    const int nthrd=cnine::roundup(std::max(n,list.max_nix()+1),32);
    Ptensors1_reduce0n_kernel<<<list.size(),nthrd,(list.max_nix()+1)*4,stream>>>
      (R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),n);
  }



  void Ptensors1_reduce1_cu(cnine::RtensorPackB& R, const cnine::RtensorPackB& x, 
    int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(R.size()==0) return;
    Ptensors1_reduce1_kernel<<<R.size(),n,0,stream>>>(R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev));
  }

  void Ptensors1_reduce1_cu(cnine::RtensorPackB& R, const cnine::RtensorPackB& x, const AindexPack& list, 
    int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(list.size()==0) return;
    const_cast<AindexPack&>(list).to_device(1);
    PTENS_ASSRT(list.dev==1);
    const int nthrd=cnine::roundup(std::max(n,list.max_nix()+1),32);
    Ptensors1_reduce1_kernel<<<list.size(),nthrd,(list.max_nix()+1)*4,stream>>>
      (R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),n);
  }



  void Ptensors1_broadcast0_cu(cnine::RtensorPackB& x, const cnine::RtensorPackB& R, 
    const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(R.size()==0) return;
    int n=R.nc; //dim_of(0,0);
    Ptensors1_broadcast0_kernel<<<R.size(),n,0,stream>>>(x.arrg+offs,x.dir.garr(dev),R.arrg,R.dir.garr(dev));
  }

  void Ptensors1_broadcast0n_cu(cnine::RtensorPackB& x, const cnine::RtensorPackB& R, 
    const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(R.size()==0) return;
    int n=R.nc; //dim_of(0,0);
    Ptensors1_broadcast0n_kernel<<<R.size(),n,0,stream>>>(x.arrg+offs,x.dir.garr(dev),R.arrg,R.dir.garr(dev));
  }

  void Ptensors1_broadcast0_cu(cnine::RtensorPackB& x, const cnine::RtensorPackB& R, const AindexPack& list, 
    const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(list.get_bmap().n==0) return;
    const_cast<AindexPack&>(list).to_device(1);
    PTENS_ASSRT(list.dev==1);
    int n=cnine::roundup(std::max(R.nc/*dim_of(0,0)*/,list.max_nix()+1),32);
    Ptensors1_broadcast0_kernel<<<list.get_bmap().n,n,(list.max_nix()+1)*4,stream>>> 
      (x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),R.arrg,R.dir.garr(dev),list.get_barr(1)); // 32 or 128
  }

  void Ptensors1_broadcast0n_cu(cnine::RtensorPackB& x, const cnine::RtensorPackB& R, const AindexPack& list, 
    const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(list.get_bmap().n==0) return;
    const_cast<AindexPack&>(list).to_device(1);
    PTENS_ASSRT(list.dev==1);
    int n=cnine::roundup(std::max(R.nc/*dim_of(0,0)*/,list.max_nix()+1),32);
    Ptensors1_broadcast0n_kernel<<<list.get_bmap().n,n,(list.max_nix()+1)*4,stream>>> 
      (x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),R.arrg,R.dir.garr(dev),list.get_barr(1)); // 32 or 128
  }



  void Ptensors1_broadcast1_cu(cnine::RtensorPackB& x, const cnine::RtensorPackB& R, 
    const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(R.size()==0) return;
    int n=R.nc; //dim_of(0,1);
    Ptensors1_broadcast1_kernel<<<R.size(),n,0,stream>>>(x.arrg+offs,x.dir.garr(dev),R.arrg,R.dir.garr(dev));
  }

  void Ptensors1_broadcast1_cu(cnine::RtensorPackB& x, const cnine::RtensorPackB& R, const AindexPack& list, 
    const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    if(list.get_bmap().n==0) return;
    const_cast<AindexPack&>(list).to_device(1);
    PTENS_ASSRT(list.dev==1);
    int n=cnine::roundup(std::max(R.nc/*dim_of(0,1)*/,list.max_nix()+1),32); // here??
    Ptensors1_broadcast1_kernel<<<list.get_bmap().n,n,(list.max_nix()+1)*4,stream>>>
      (x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),R.arrg,R.dir.garr(dev),list.get_barr(1));
  }



  void Ptensors1_add_outer10_cu(cnine::RtensorPackB& r, const cnine::RtensorPackB& x, const cnine::RtensorPackB& y, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    if(r.size()==0) return;
    dim3 threads(x.dim_of(0,1),y.dim_of(0,0));
    Ptensors1_add_outer10_kernel<<<r.size(),threads,0,stream>>>
      (r.arrg,r.dir.garr(dev),x.arrg,x.dir.garr(dev),y.arrg,y.dir.garr(dev));
  }

  void Ptensors1_add_outer10_back0_cu(cnine::RtensorPackB& x, const cnine::RtensorPackB& r, const cnine::RtensorPackB& y, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    if(r.size()==0) return;
    Ptensors1_add_outer10_back0_kernel<<<r.size(),x.dim_of(0,1),0,stream>>>
      (x.arrg,x.dir.garr(dev),r.arrg,r.dir.garr(dev),y.arrg,y.dir.garr(dev));
  }

  void Ptensors1_add_outer10_back1_cu(cnine::RtensorPackB& y, const cnine::RtensorPackB& r, const cnine::RtensorPackB& x, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    if(r.size()==0) return;
    Ptensors1_add_outer10_back1_kernel<<<r.size(),y.dim_of(0,0),0,stream>>>
      (y.arrg,y.dir.garr(dev),r.arrg,r.dir.garr(dev),x.arrg,x.dir.garr(dev));
  }



  void Ptensors1_add_outer01_cu(cnine::RtensorPackB& r, const cnine::RtensorPackB& x, const cnine::RtensorPackB& y, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    if(r.size()==0) return;
    dim3 threads(x.dim_of(0,0),y.dim_of(0,1));
    Ptensors1_add_outer01_kernel<<<r.size(),threads,0,stream>>>
      (r.arrg,r.dir.garr(dev),x.arrg,x.dir.garr(dev),y.arrg,y.dir.garr(dev));
  }

  void Ptensors1_add_outer01_back0_cu(cnine::RtensorPackB& x, const cnine::RtensorPackB& r, const cnine::RtensorPackB& y, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    if(r.size()==0) return;
    Ptensors1_add_outer01_back0_kernel<<<r.size(),x.dim_of(0,0),0,stream>>>
      (x.arrg,x.dir.garr(dev),r.arrg,r.dir.garr(dev),y.arrg,y.dir.garr(dev));
  }

  void Ptensors1_add_outer01_back1_cu(cnine::RtensorPackB& y, const cnine::RtensorPackB& r, const cnine::RtensorPackB& x, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    if(r.size()==0) return;
    Ptensors1_add_outer01_back1_kernel<<<r.size(),y.dim_of(0,1),0,stream>>>
      (y.arrg,y.dir.garr(dev),r.arrg,r.dir.garr(dev),x.arrg,x.dir.garr(dev));
  }


}


#endif 
