#include "hip/hip_runtime.h"
#ifndef _Ptensors0_cu
#define _Ptensors0_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/tuple.h>

#include "Ptens_base.hpp"
#include "RtensorPack.hpp"
#include "AindexPack.hpp"
#include "CSRmatrix.hpp"
//#include "Ptensors0.hpp"
//#include "Rtensor2_view.hpp"
//#include "Rtensor3_view.hpp"
//#include "Itensor1_view.hpp"
//#include "Itensor2_view.hpp"
//#include "CUDAhelpers.hpp"


__global__ void Ptensors0_reduce0_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;
  rarr[rdir[2*i]+c]+=xarr[xdir[2*i]+c];
}


__global__ void Ptensors0_reduce0_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const int* xiarr, const int* xidir, const int n){
  const int b=blockIdx.x;
  const int c=threadIdx.x;
  const int src=xiarr[xidir[2*b]];
  rarr[rdir[2*b]+c]+=xarr[xdir[2*src]+c];
}


__global__ void Ptensors0_broadcast0_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;
  xarr[xdir[2*i]+c]+=rarr[rdir[2*i]+c];
}


__global__ void Ptensors0_broadcast0_kernel(float* xarr, const int* xdir, const int* xiarr, const int* xidir, const float* rarr, const int* rdir, const int* bmap){
  const int b=blockIdx.x;
  const int c=threadIdx.x;
  const int boffs=bmap[3*b];
  const int N=bmap[3*b+1];
  const int target=bmap[3*b+2];

  float t=0;
  for(int j=0; j<N; j++){
    const int src=bmap[boffs+2*j];
    const float w=*reinterpret_cast<const float*>(bmap+boffs+2*j+1);
    t+=w*rarr[rdir[2*src]+c];
  }
  xarr[xdir[2*target]+c]+=t;
}


__global__ void Ptensors0_gather_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const float* marr, const int* mdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;

  const int moffs=mdir[2*i];
  const int N=mdir[2*i+1]/2;
  float t=0;
  for(int j=0; j<N; j++){
    const int jix=*reinterpret_cast<const int*>(marr+moffs+2*j);
    const int jweight=marr[moffs+2*j+1];
    t+=jweight*xarr[xdir[2*jix]+c];
  }
  rarr[rdir[2*i]+c]+=t;
}


// ---- Outer -----------------------------------------------------------------------------------------------


__global__ void Ptensors0_add_outer00_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const float* yarr, const int* ydir){
  const int q=blockIdx.x;
  const int xc=threadIdx.x;
  const int yc=threadIdx.y;
  const int rc=xc*ydir[2*q+1]+yc;
  const int nxc=xdir[2*q+1];
  const int nrc=rdir[2*q+1];

  rarr[rdir[2*q]+rc]+=yarr[ydir[2*q]+yc]+t*xarr[xdir[2*q]+xc];
}


__global__ void Ptensors0_add_outer00_back0_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir, const float* yarr, const int* ydir){
  const int q=blockIdx.x;
  const int xc=threadIdx.x;
  const int rc=xc*ydir[2*q+1];
  //const int nxc=xdir[2*q+1];
  const int nyc=ydir[2*q+1];
  //const int nrc=rdir[2*q+1];

  const float* r=rarr+rdir[2*q]+rc;
  const float* y=yarr+ydir[2*q];

  float t=0;
  for(int yc=0; yc<nyc; yc++)
    t+=r[yc]*y[yc];
  xarr[xdir[2*q]+xc]+=t;
}


__global__ void Ptensors0_add_outer00_back1_kernel(float* yarr, const int* ydir, const float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int q=blockIdx.x;
  const int yc=threadIdx.x;
  const int nxc=xdir[1*q+1];
  const int nyc=ydir[2*q+1];
  //const int nrc=rdir[2*q+1];

  float t=0;
  const float* x=xarr+xdir[3*q];
  const float* r=rarr+rdir[3*q]+yc;
  for(int xc=0; xc<nxc; xc++)
    t+=r[nyc*xc]*x[xc];
  yarr[ydir[2*q]+yc]+=t;
}


// -----------------------------------------------------------------------------------------------------------


namespace ptens{

  void Ptensors0_reduce0_cu(cnine::RtensorPack& R, const cnine::RtensorPack& x, int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    Ptensors0_reduce0_kernel<<<R.size(),n,0,stream>>>(R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev));
  }

  void Ptensors0_reduce0_cu(cnine::RtensorPack& R, const cnine::RtensorPack& x, const AindexPack& list, int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    const_cast<AindexPack&>(list).to_device(1);
    Ptensors0_reduce0_kernel<<<R.size(),n,0,stream>>>(R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),n);
  }

  void Ptensors0_broadcast0_cu(cnine::RtensorPack& x, const cnine::RtensorPack& R, const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    Ptensors0_broadcast0_kernel<<<R.size(),x.dim_of(0,0),0,stream>>>
      (x.arrg+offs,x.dir.garr(dev),R.arrg,R.dir.garr(dev));
  }

  void Ptensors0_broadcast0_cu(cnine::RtensorPack& x, const cnine::RtensorPack& R, const AindexPack& list, const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    const_cast<AindexPack&>(list).to_device(1);
    Ptensors0_broadcast0_kernel<<<R.size(),R.dim_of(0,0),0,stream>>>
      (x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),R.arrg,R.dir.garr(dev),list.get_barr(1));
  }

  void Ptensors0_gather_cu(cnine::RtensorPack& r, const cnine::RtensorPack& x, const cnine::CSRmatrix<float>& gmap, const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    const_cast<cnine::CSRmatrix<float>&>(gmap).to_device(dev);
    Ptensors0_gather_kernel<<<r.size(),x.dim_of(0,0),0,stream>>>
      (r.arrg,r.dir.garr(dev),x.arrg,x.dir.garr(dev),gmap.arrg,gmap.dir.garr(dev));
  }


  void Ptensors0_add_outer_cu(cnine::RtensorPack& r, const cnine::RtensorPack& x, const cnine::RtensorPack& y, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    dim3 threads(x.dim_of(0,0),y.dim_of(0,0));
    Ptensors0_add_outer_kernel<<<r.size(),threads,0,stream>>>
      (r.arrg,r.dir.garr(dev),x.arrg,x.dir.garr(dev),y.arrg,y.dir.garr(dev));
  }

  void Ptensors0_add_outer_back0_cu(cnine::RtensorPack& x, const cnine::RtensorPack& r, const cnine::RtensorPack& y, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    Ptensors0_add_outer_back0_kernel<<<r.size(),x.dim_of(0,0),0,stream>>>
      (x.arrg,x.dir.garr(dev),r.arrg,r.dir.garr(dev),y.arrg,y.dir.garr(dev));
  }

  void Ptensors0_add_outer_back1_cu(cnine::RtensorPack& y, const cnine::RtensorPack& r, const cnine::RtensorPack& x, 
    const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(y.dev==1);
    Ptensors0_add_outer_back1_kernel<<<r.size(),y.dim_of(0,0),0,stream>>>
      (y.arrg,y.dir.garr(dev),r.arrg,r.dir.garr(dev),x.arrg,x.dir.garr(dev));
  }


}

#endif 
